#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <math.h>
#include "utils.h"
// Assertion to check for errors
#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"CUDA_SAFE_CALL: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

#define NUM_THREADS_PER_BLOCK 	5000	
#define NUM_BLOCKS 				1	
#define PRINT_TIME 				1
#define TOL						1e-6
#define TEST_CASE				1
#define X_DIM                   6
#define X_LENGTH                5000
#define ETA   					1.0
#define MAX_ITERS		10000
#define IMUL(a, b) __mul24(a, b)

__global__ void calculate_weights(data_t* X, char* Y, data_t* W, char* misclassified,int x_length, int x_dim, double eta){
	__shared__ data_t block_weights[NUM_THREADS_PER_BLOCK][X_DIM]; // 500 x 6
	int tx = threadIdx.x;

	int i,j;
	
		if(misclassified[tx] == 1){
			for(j= 0; j < x_dim;j++){
				block_weights[tx][j] = eta*X[tx*x_dim+j]*Y[tx];
			}	
		}
		else{
			for(j=0; j < x_dim; j++){
				block_weights[tx][j] = 0;
			}
		}
	__syncthreads();

	float sum;
	if(tx==399){
		for(j=0;j<x_dim;j++){
			sum = 0;
			for(i=0; i < NUM_THREADS_PER_BLOCK;i++){
				sum = sum+  block_weights[i][j];
			}
			W[j]+= sum;
		}
	}
}

__global__ void classify(data_t* X, char* Y, data_t* W, char* misclassified, char* not_classified, int* sum_missed,  int x_dim){
	__shared__  data_t score_shared[NUM_THREADS_PER_BLOCK];
	int tx = threadIdx.x;
	int j;
	score_shared[tx] =0;
	sum_missed[tx] = 0;
	not_classified[tx] = 0;
	for(j=0;j < x_dim; ++j){
		score_shared[tx] += X[tx*x_dim + j]*W[j];
	}
	__syncthreads();
	misclassified[tx] = score_shared[tx]*Y[tx] <= 0.0 ? 1:0;
	__syncthreads();
	if(misclassified[tx] == 1){
		sum_missed[tx] = 1;	
		not_classified[tx] = 1;
	}
	__syncthreads();
}


int main(int argc, char **argv){
	// GPU Timing variables
	hipEvent_t start, stop;
	float elapsed_gpu;
	int test_case = TEST_CASE;
	int h_x_length = X_LENGTH;
	int h_x_dim = X_DIM;
	int line_counter = 0;
	int i;
	char not_classified = 1;
	int iters = 0;


	// global variables on GPU
	data_t* g_W;			
	data_t* g_X;
	float* g_score;
	char* g_Y;
	char* g_not_classified;
	char* g_misclassified;
	int* g_sum_missed;


	//global arrays on host
	float* h_W;
	float* h_X;
	char* h_Y;
	float* h_score;
	char* h_misclassified;	
	char* h_not_classified;
	int* h_sum_missed;
	int missed = 0;

	 // Select GPU
	CUDA_SAFE_CALL(hipSetDevice(1));

	// Allocate GPU memory
	size_t allocSize_X =  h_x_dim * h_x_length * sizeof(data_t);
	size_t allocSize_Y = h_x_length * sizeof(char);
	size_t allocSize_W = h_x_dim * sizeof(data_t);
	size_t allocSize_Score = h_x_length * sizeof(float);
	size_t allocSize_sumMissed = sizeof(int)*h_x_length;

	CUDA_SAFE_CALL(hipMalloc((void **)&g_W, allocSize_W))
	CUDA_SAFE_CALL(hipMalloc((void **)&g_X, allocSize_X));
	CUDA_SAFE_CALL(hipMalloc((void **)&g_Y, allocSize_Y));
	CUDA_SAFE_CALL(hipMalloc((void **)&g_score, allocSize_Score));
	CUDA_SAFE_CALL(hipMalloc((void **)&g_misclassified, allocSize_Y));	
	CUDA_SAFE_CALL(hipMalloc((void **)&g_sum_missed, allocSize_sumMissed));
	CUDA_SAFE_CALL(hipMalloc((void **)&g_not_classified, allocSize_Y));

	// Allocate arrays on host memory
	h_X                     = (data_t *) malloc(allocSize_X);
	h_Y                   	= (char *) malloc(allocSize_Y);
	h_W              	= (data_t *) malloc(allocSize_W);
	h_misclassified 	= (char *) malloc(allocSize_Y);
	h_score			= (float *) malloc(allocSize_Score);
	h_sum_missed		= (int *) malloc(allocSize_sumMissed);
	h_not_classified 	= (char *) malloc(allocSize_Y);

	for(i=0;i< h_x_length;i++){
		h_misclassified[i] = 1;
	}
	for(i=0; i < h_x_dim; i++){
		h_W[i] = 0;
	}

    FILE* stream = fopen("data.csv", "r");

    char line[1024];
    while (fgets(line, 1024, stream))
    {
        char* tmp = strdup(line);
        int idx = line_counter*h_x_dim;
        h_X[idx] = 1.0;
        h_X[idx + 1] = strtod(getfield(tmp, 1), NULL);
        tmp = strdup(line);
        h_X[idx + 2] = strtod(getfield(tmp, 2), NULL);
        h_X[idx + 3] = h_X[idx + 1]*h_X[idx + 2]; // xy
        h_X[idx + 4] = h_X[idx + 1]*h_X[idx + 1]; // x^2 
        h_X[idx + 5] = h_X[idx + 2]*h_X[idx + 2]; // y^2
        // NOTE strtok clobbers tmp
        free(tmp);
        line_counter++;
    }

    assign_labels(h_X, h_x_length, h_x_dim, test_case, h_Y);
/*
    for(i=0; i < h_x_length; ++i){ 
        switch(test_case) {
            case 1:
                h_Y[i] = (0.2*(h_X[i*h_x_dim + 0] - 0.5)) +
                    (.6-h_X[i*h_x_dim + 1]) > 0 ? 1 : -1;
                break;
            case 2:
                h_Y[i] = (h_X[i*h_x_dim + 0]-.5)*(h_X[i*h_x_dim + 0]-.5) +
                    (h_X[i*h_x_dim + 1]-.5)*(h_X[i*h_x_dim + 1]-.5) > 0.09 ? 1 : -1;
                break;
            case 3:
                h_Y[i] = 4*(h_X[i*h_x_dim + 0]-.5)*4*(h_X[i*h_x_dim + 0]-.5) +
                    (.2-h_X[i*h_x_dim + 1]) > 0 ? 1 : -1;
                break;
            default:
                h_Y[i] = 0;
        }
    }*/
 }


    // Transfer the arrays to the GPU memory
	CUDA_SAFE_CALL(hipMemcpy(g_X, h_X, allocSize_X, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(g_Y, h_Y, allocSize_Y, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(g_W, h_W, allocSize_W, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(g_misclassified, h_misclassified, allocSize_Y, hipMemcpyHostToDevice));

#ifdef PRINT_TIME
hipEventCreate(&start);
hipEventCreate(&stop);
hipEventRecord(start,0);
#endif

while(not_classified && iters <= MAX_ITERS){
		
		// Increment iters
		iters++;
		printf("iters: %d \n", iters);
		// Set condition to zero (to avoid infinite while loop) and set it to one if there's an element that is misclassified
		not_classified = 0;
		// One block with 500 threads (one thread working on each row of data in X)
		calculate_weights<<<1,500>>>(g_X, g_Y,g_W,g_misclassified,500, 6, 1);
		CUDA_SAFE_CALL(hipPeekAtLastError());
		// Copy weight vector to host
		CUDA_SAFE_CALL(hipMemcpy(h_W, g_W, allocSize_W, hipMemcpyDeviceToHost));

		// Check classification success		
		classify<<<1,500>>>(g_X, g_Y, g_W, g_misclassified, g_not_classified, g_sum_missed,6);
		CUDA_SAFE_CALL(hipPeekAtLastError());

		// Copy arrays back to host
		CUDA_SAFE_CALL(hipMemcpy(h_not_classified, g_not_classified,allocSize_Y, hipMemcpyDeviceToHost));
		CUDA_SAFE_CALL(hipMemcpy(h_sum_missed, g_sum_missed,allocSize_sumMissed, hipMemcpyDeviceToHost));
	//	CUDA_SAFE_CALL(hipMemcpy(h_misclassified, g_misclassified,allocSize_Y, hipMemcpyDeviceToHost));

		for(i=0;i<h_x_length;i++){
			not_classified += h_not_classified[i];		
		}
}

#ifdef PRINT_TIME
hipEventRecord(stop,0);
hipEventSynchronize(stop);
hipEventElapsedTime(&elapsed_gpu, start, stop);
printf("GPU execution time: %f (msec) \n", elapsed_gpu);
hipEventDestroy(start);
#endif


	printf("\n");
	for(i=0;i < h_x_length; ++i){
		missed += h_sum_missed[i];
		printf("%d ", h_sum_missed[i]);	
	}
	printf("\nIterations: %d", iters);
	if(missed == 0){
		printf("Perfectly separated data\n");
	}
	else{
		printf("Finished MAX_ITERS and still %d misclassified \n", missed);
	}

		// Free-up device and host memory
	CUDA_SAFE_CALL(hipFree(g_X));
	CUDA_SAFE_CALL(hipFree(g_Y));
	CUDA_SAFE_CALL(hipFree(g_W));
	CUDA_SAFE_CALL(hipFree(g_sum_missed));
	CUDA_SAFE_CALL(hipFree(g_not_classified));
	CUDA_SAFE_CALL(hipFree(g_score));
	CUDA_SAFE_CALL(hipFree(g_misclassified));		   
	free(h_X);
	free(h_not_classified);
	free(h_sum_missed);
	free(h_Y);
	free(h_W);
	free(h_misclassified);
	free(h_score);
	return 0;
}


