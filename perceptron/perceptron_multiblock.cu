#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <math.h>
// Assertion to check for errors
#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"CUDA_SAFE_CALL: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

#define NUM_THREADS_PER_BLOCK 	500	
#define NUM_BLOCKS 		20	
#define PRINT_TIME		1
#define TEST_CASE		3
#define X_DIM                   6
#define X_LENGTH                10000
#define START_ETA		0.1
#define ETA   			1.0
#define DELTA			.05
#define MAX_ITERS		50000
#define IMUL(a, b) __mul24(a, b)


const char* getfield(char* line, int num);

__global__ void calculate_weights(float* X, char* Y, float* W, char* misclassified,int x_length, int x_dim, double eta){
	__shared__ float block_weights[NUM_THREADS_PER_BLOCK][X_DIM]; // 500 x 6
	int tx = threadIdx.x;
	int tx_global = blockIdx.x*blockDim.x + threadIdx.x;
	int i,j;
	
		if(misclassified[tx_global] == 1){
			for(j= 0; j < x_dim;j++){
				block_weights[tx][j] = eta*X[tx_global*x_dim+j]*Y[tx_global];
			}	
		}
		else{
			for(j=0; j < x_dim; j++){
				block_weights[tx][j] = 0;
			}
		}
	__syncthreads();

	float sum;
	if(tx==399){
		for(j=0;j<x_dim;j++){
			sum = 0;
			for(i=0; i < NUM_THREADS_PER_BLOCK;i++){
				sum = sum+  block_weights[i][j];
			}
			//W_mat[blockIdx.x*x_dim+j]= sum;
			W[j] += sum;
		}
	}
}

__global__ void classify(float* X, char* Y, float* W, char* misclassified, int* not_classified, int* sum_missed,  int x_dim){
	__shared__  float score_shared[NUM_THREADS_PER_BLOCK];
	__shared__ int not_classified_shared[NUM_THREADS_PER_BLOCK];
	int tx = threadIdx.x;
	int tx_global = blockIdx.x*blockDim.x + threadIdx.x;
	int j;
	score_shared[tx] =0;
	sum_missed[tx_global] = 0;
	not_classified[tx_global] = 0;
	__syncthreads();
	not_classified_shared[tx] = 0;
	for(j=0;j < x_dim; ++j){
		score_shared[tx] += X[tx_global*x_dim + j]*W[j];
	}
	__syncthreads();
	misclassified[tx_global] = score_shared[tx]*Y[tx_global] <= 0.0 ? 1:0;
	if(misclassified[tx_global] == 1){
		sum_missed[tx_global] = 1;	
		not_classified[tx_global] = 1;
	}
	/*if(tx == 399){
		int notClassified = 0;
		for(j=0; j < NUM_THREADS_PER_BLOCK; j++){
			if(not_classified_shared[j] == 1){
				notClassified = 1;
			}
		}
		if(notClassified){
			not_classified[blockIdx.x] = 1;
		}
	 }*/
}


int main(int argc, char **argv){
	// GPU Timing variables
	hipEvent_t start, stop;
	float elapsed_gpu;
	int test_case = TEST_CASE;
	int h_x_length = X_LENGTH;
	int h_x_dim = X_DIM;
	int line_counter = 0;
	int i;
	int not_classified = 1;
	int iters = 0;
	float eta = ETA;
	float start_eta = START_ETA;
	float delta = DELTA;
	int sum_missed_iters[19][2];
	// global variables on GPU
	float* g_W;			
	float* g_X;
	float* g_score;
	char* g_Y;
	int* g_not_classified;
	char* g_misclassified;
	int* g_sum_missed;
	float* g_W_matrix;

	//global arrays on host
	float* h_W;
	float* h_W_matrix;
	float* h_X;
	float* h_score;
	char* h_misclassified;	
	char* h_Y;
	int* h_not_classified;
	int* h_sum_missed;
	int missed = 0;

	 // Select GPU
	CUDA_SAFE_CALL(hipSetDevice(1));

	// Allocate GPU memory
	size_t allocSize_X =  h_x_dim * h_x_length * sizeof(float);
	size_t allocSize_Y = h_x_length * sizeof(char);
	size_t allocSize_W = h_x_dim * sizeof(float);
	size_t allocSize_Score = h_x_length * sizeof(float);
	size_t allocSize_sumMissed = sizeof(int)*h_x_length;
	size_t allocSize_notClassified = sizeof(int) * NUM_BLOCKS;
	size_t allocSize_W_mat = sizeof(float)*h_x_dim*NUM_BLOCKS;

	CUDA_SAFE_CALL(hipMalloc((void **)&g_W, allocSize_W))
	CUDA_SAFE_CALL(hipMalloc((void **)&g_X, allocSize_X));
	CUDA_SAFE_CALL(hipMalloc((void **)&g_Y, allocSize_Y));
	CUDA_SAFE_CALL(hipMalloc((void **)&g_score, allocSize_Score));
	CUDA_SAFE_CALL(hipMalloc((void **)&g_misclassified, allocSize_Y));	
	CUDA_SAFE_CALL(hipMalloc((void **)&g_sum_missed, allocSize_sumMissed));
	CUDA_SAFE_CALL(hipMalloc((void **)&g_not_classified, allocSize_sumMissed));
 	CUDA_SAFE_CALL(hipMalloc((void **)&g_W_matrix, allocSize_W_mat));
	// Allocate arrays on host memory
	h_X                     = (float *) malloc(allocSize_X);
	h_Y                   	= (char *) malloc(allocSize_Y);
	h_W              	= (float *) malloc(allocSize_W);
	h_W_matrix		= (float *) malloc(allocSize_W_mat);
	h_misclassified 	= (char *) malloc(allocSize_Y);
	h_score			= (float *) malloc(allocSize_Score);
	h_sum_missed		= (int *) malloc(allocSize_sumMissed);
	h_not_classified 	= (int *) malloc(allocSize_sumMissed);

	for(i=0;i< h_x_length;i++){
		h_misclassified[i] = 1;
	}
	for(i=0; i < h_x_dim; i++){
		h_W[i] = 0;
	}
	/*int h_W_size = NUM_BLOCKS* h_x_dim;
	for(i = 0; i < h_W_size; i++){
		h_W_matrix[i] = 0;
	}*/

    FILE* stream = fopen("data.csv", "r");

    char line[1024];
    while (fgets(line, 1024, stream))
    {
        char* tmp = strdup(line);
        int idx = line_counter*h_x_dim;
        h_X[idx] = 1.0;
        h_X[idx + 1] = strtod(getfield(tmp, 1), NULL);
        tmp = strdup(line);
        h_X[idx + 2] = strtod(getfield(tmp, 2), NULL);
        h_X[idx + 3] = h_X[idx + 1]*h_X[idx + 2]; // xy
        h_X[idx + 4] = h_X[idx + 1]*h_X[idx + 1]; // x^2 
        h_X[idx + 5] = h_X[idx + 2]*h_X[idx + 2]; // y^2
        // NOTE strtok clobbers tmp
        free(tmp);
        line_counter++;
    }

	printf("x length by line counter: %d \n", line_counter);

    for(i=0; i < h_x_length; ++i){ 
        switch(test_case) {
            case 1:
                h_Y[i] = (0.2*(h_X[i*h_x_dim + 1] - 0.5)) +
                    (.6-h_X[i*h_x_dim + 2]) > 0 ? 1 : -1;
                break;
            case 2:
                h_Y[i] = (h_X[i*h_x_dim + 1]-.5)*(h_X[i*h_x_dim + 1]-.5) +
                    (h_X[i*h_x_dim + 2]-.5)*(h_X[i*h_x_dim + 2]-.5) > 0.09 ? 1 : -1;
                break;
            case 3:
                h_Y[i] = 4*(h_X[i*h_x_dim + 1]-.5)*(h_X[i*h_x_dim + 1]-.5) +
                    (.2-h_X[i*h_x_dim + 2]) > 0 ? 1 : -1;
                break;
            default:
                h_Y[i] = 0;
        }
    }
float exec_times[19][2];
int k, j, acc;
int index = 0;
float current_eta = start_eta;
for(k = 0; k < 19; k++){
	for(i=0;i< h_x_length;i++){
		h_misclassified[i] = 1;
	}
	for(i=0; i < h_x_dim; i++){
		h_W[i] = 0;
	}
/*	int h_W_size = NUM_BLOCKS* h_x_dim;
	for(i = 0; i < h_W_size; i++){
		h_W_matrix[i] = 0;
	}*/
	for(i=0; i < NUM_BLOCKS; i++){
		h_not_classified[i] = 0;
	}
	CUDA_SAFE_CALL(hipMemcpy(g_X, h_X, allocSize_X, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(g_Y, h_Y, allocSize_Y, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(g_W, h_W, allocSize_W, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(g_misclassified, h_misclassified, allocSize_Y, hipMemcpyHostToDevice));
//	CUDA_SAFE_CALL(hipMemcpy(g_W_matrix, h_W_matrix, allocSize_W_mat, hipMemcpyHostToDevice));
	iters = 0;
	missed = 0;
	not_classified = 1;
	#ifdef PRINT_TIME
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	#endif
//	int max_iters = MAX_ITERS;

dim3 dimBlock(NUM_THREADS_PER_BLOCK, 1, 1);
dim3 dimGrid(NUM_BLOCKS, 1);
while(not_classified && iters <= MAX_ITERS){
		// Increment iters
		iters++;
		// Set condition to zero (to avoid infinite while loop) and set it to one if there's an element that is misclassified
		not_classified = 0;
		// One block with 500 threads (one thread working on each row of data in X)
		calculate_weights<<<dimGrid, dimBlock>>>(g_X, g_Y,g_W,g_misclassified,h_x_length, h_x_dim, current_eta);
		CUDA_SAFE_CALL(hipPeekAtLastError());
		hipDeviceSynchronize();
		// Copy weight vector to host
	//	CUDA_SAFE_CALL(hipMemcpy(h_W_matrix, g_W_matrix, allocSize_W_mat, hipMemcpyDeviceToHost));
	/*	CUDA_SAFE_CALL(hipMemcpy(h_W, g_W, allocSize_W, hipMemcpyDeviceToHost));
		printf("Weight vector: ");
		for(i=0 ; i < h_x_dim; i++){
			printf("%f", h_W[i]);
		}
		printf("\n");*/
	//	printf("Weight vector after accumulating matrix values \n");
	/*	for(i=0;i<h_x_dim;i++){
			acc =0;
			for(j=0;j<NUM_BLOCKS;j++){
				acc += h_W_matrix[j*h_x_dim + i];
			}
			h_W[i] += acc;
			printf("%f ", h_W[i]);
		}
		printf("\n");
	*/
	//	CUDA_SAFE_CALL(hipMemcpy(g_W, h_W, allocSize_W, hipMemcpyHostToDevice));
		// Check classification success	
	//	hipDeviceSynchronize();	
		classify<<<dimGrid, dimBlock>>>(g_X, g_Y, g_W, g_misclassified, g_not_classified, g_sum_missed,h_x_dim);
		CUDA_SAFE_CALL(hipPeekAtLastError());
		hipDeviceSynchronize();
		// Copy arrays back to host
		CUDA_SAFE_CALL(hipMemcpy(h_not_classified, g_not_classified,allocSize_sumMissed, hipMemcpyDeviceToHost));
		CUDA_SAFE_CALL(hipMemcpy(h_sum_missed, g_sum_missed,allocSize_sumMissed, hipMemcpyDeviceToHost));
	//	printf("not classified before acc: %d \n", not_classified);
	//	hipDeviceSynchronize();
		for(i=0;i<h_x_length;i++){
			not_classified += h_not_classified[i];
			//printf("%d ", h_not_classified[i]);		
		}
		printf("not classified after acc: %d \n", not_classified);
}
	

	#ifdef PRINT_TIME
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_gpu, start, stop);
	exec_times[index][1] = elapsed_gpu;
	exec_times[index][0] = current_eta;
	hipEventDestroy(start);
	#endif
	printf("\n");
		for(i=0;i < h_x_length; ++i){
			missed += h_sum_missed[i];
		}
		printf("current_eta: %f, eta: %f, start_eta: %f, index: %d, delta: %f \n", current_eta, eta, start_eta, index, delta);
		if(missed == 0){
			printf("Perfectly separated data\n");
		}
		else{
			printf("Finished MAX_ITERS (%d iters) and still %d misclassified \n", iters, missed);
		}
	sum_missed_iters[index][0] = missed;
	sum_missed_iters[index][1] = iters;
	printf("iters out of loop: %d", iters);
	current_eta += delta;
	index++;
}
	printf("Iters		Exec time (ms)		Sum Missed: 		Iters: \n");
	for(i=0;i<19; i++){
		printf("\t%f\t\t\t%f\t\t%d\t\t\t%d\n", exec_times[i][0], exec_times[i][1], sum_missed_iters[i][0], sum_missed_iters[i][1]);
	}
		// Free-up device and host memory
	CUDA_SAFE_CALL(hipFree(g_X));
	CUDA_SAFE_CALL(hipFree(g_Y));
	CUDA_SAFE_CALL(hipFree(g_W));
	CUDA_SAFE_CALL(hipFree(g_sum_missed));
	CUDA_SAFE_CALL(hipFree(g_not_classified));
	CUDA_SAFE_CALL(hipFree(g_score));
	CUDA_SAFE_CALL(hipFree(g_misclassified));		   
	free(h_X);
	free(h_not_classified);
	free(h_sum_missed);
	free(h_Y);
	free(h_W);
	free(h_misclassified);
	free(h_score);
	return 0;
}

const char* getfield(char* line, int num) {
    const char* tok;
    for (tok = strtok(line, ",");
            tok && *tok;
            tok = strtok(NULL, ",\n"))
    {
        if (!--num)
            return tok;
    }
    return NULL;
}
