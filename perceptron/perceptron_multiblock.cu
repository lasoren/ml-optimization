
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <math.h>
// Assertion to check for errors
#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"CUDA_SAFE_CALL: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

#define NUM_THREADS_PER_BLOCK 	300	
#define NUM_BLOCKS 		20	
#define PRINT_TIME		1
#define TEST_CASE		3
#define X_DIM                   6
#define X_LENGTH                6000
#define START_ETA		0.1
#define ETA   			1.0
#define DELTA			.05
#define MAX_ITERS		40000
#define IMUL(a, b) __mul24(a, b)


const char* getfield(char* line, int num);

__global__ void calculate_weights(float* X, char* Y, float* W, char* misclassified,int x_length, int x_dim, double eta){
	__shared__ float block_weights[NUM_THREADS_PER_BLOCK][X_DIM];
	int tx = threadIdx.x;
	int tx_global = blockIdx.x*blockDim.x + threadIdx.x;
	int block_id = blockIdx.x;
	int i,j;
	
		if(misclassified[tx_global] == 1){
			for(j= 0; j < x_dim;j++){
				block_weights[tx][j] = eta*X[tx_global*x_dim+j]*Y[tx_global];
			}	
		}
		else{
			for(j=0; j < x_dim; j++){
				block_weights[tx][j] = 0;
			}
		}
	__syncthreads();

	float sum;
	if(tx==1){
		for(j=0;j<x_dim;j++){
			sum = 0;
			for(i=0; i < NUM_THREADS_PER_BLOCK;i++){
				sum = sum+  block_weights[i][j];
			}
			W[block_id*x_dim+j]= sum;
		}
	}
}

__global__ void classify(float* X, char* Y, float* W, char* misclassified, int* not_classified, int* sum_missed,  int x_dim){
	float score;
	int tx = threadIdx.x;
	int tx_global = blockIdx.x*blockDim.x + threadIdx.x;
	int j;
	score = 0;
	sum_missed[tx_global] = 0;
	not_classified[tx_global] = 0;
	__syncthreads();
	for(j=0;j < x_dim; j++){
		score += X[tx_global*x_dim + j]*W[j];
	}
	misclassified[tx_global] = score*Y[tx_global] <= 0.0 ? 1:0;
	if(misclassified[tx_global] == 1){
		sum_missed[tx_global] = 1;	
		not_classified[tx_global] = 1;
	}
}


int main(int argc, char **argv){
	// GPU Timing variables
	hipEvent_t start, stop;
	float elapsed_gpu;
	int test_case = TEST_CASE;
	int h_x_length = X_LENGTH;
	int h_x_dim = X_DIM;
	int line_counter = 0;
	int i;
	int total_missed =0;
	float verify = 0;
	int not_classified = 1;
	int iters = 0;
	float eta = ETA;
	float start_eta = START_ETA;
	float delta = DELTA;
	int sum_missed_iters[5][2];
	// global variables on GPU
	float* g_W;			
	float* g_X;
	float* g_score;
	char* g_Y;
	int* g_not_classified;
	char* g_misclassified;
	int* g_sum_missed;
	float* g_W_matrix;

	//global arrays on host
	float* h_W;
	float* h_W_matrix;
	float* h_X;
	float* h_score;
	char* h_misclassified;	
	char* h_Y;
	int* h_not_classified;
	int* h_sum_missed;
	int missed = 0;

	 // Select GPU
	CUDA_SAFE_CALL(hipSetDevice(1));

	// Allocate GPU memory
	size_t allocSize_X =  h_x_dim * h_x_length * sizeof(float);
	size_t allocSize_Y = h_x_length * sizeof(char);
	size_t allocSize_W = h_x_dim * sizeof(float);
	size_t allocSize_Score = h_x_length * sizeof(float);
	size_t allocSize_sumMissed = sizeof(int)*h_x_length;
	size_t allocSize_notClassified = sizeof(int) * NUM_BLOCKS;
	size_t allocSize_W_mat = sizeof(float)*h_x_dim*NUM_BLOCKS;

	CUDA_SAFE_CALL(hipMalloc((void **)&g_W, allocSize_W))
	CUDA_SAFE_CALL(hipMalloc((void **)&g_X, allocSize_X));
	CUDA_SAFE_CALL(hipMalloc((void **)&g_Y, allocSize_Y));
	CUDA_SAFE_CALL(hipMalloc((void **)&g_score, allocSize_Score));
	CUDA_SAFE_CALL(hipMalloc((void **)&g_misclassified, allocSize_Y));	
	CUDA_SAFE_CALL(hipMalloc((void **)&g_sum_missed, allocSize_sumMissed));
	CUDA_SAFE_CALL(hipMalloc((void **)&g_not_classified, allocSize_sumMissed));
 	CUDA_SAFE_CALL(hipMalloc((void **)&g_W_matrix, allocSize_W_mat));
	// Allocate arrays on host memory
	h_X                     = (float *) malloc(allocSize_X);
	h_Y                   	= (char *) malloc(allocSize_Y);
	h_W              	= (float *) malloc(allocSize_W);
	h_W_matrix		= (float *) malloc(allocSize_W_mat);
	h_misclassified 	= (char *) malloc(allocSize_Y);
	h_score			= (float *) malloc(allocSize_Score);
	h_sum_missed		= (int *) malloc(allocSize_sumMissed);
	h_not_classified 	= (int *) malloc(allocSize_sumMissed);

    FILE* stream = fopen("data.csv", "r");

    char line[1024];
    while (fgets(line, 1024, stream) && line_counter < X_LENGTH)
    {
        char* tmp = strdup(line);
        int idx = line_counter*h_x_dim;
        h_X[idx] = 1.0;
        h_X[idx + 1] = strtod(getfield(tmp, 1), NULL);
        tmp = strdup(line);
        h_X[idx + 2] = strtod(getfield(tmp, 2), NULL);
        h_X[idx + 3] = h_X[idx + 1]*h_X[idx + 2]; // xy
        h_X[idx + 4] = h_X[idx + 1]*h_X[idx + 1]; // x^2 
        h_X[idx + 5] = h_X[idx + 2]*h_X[idx + 2]; // y^2
        // NOTE strtok clobbers tmp
        free(tmp);
        line_counter++;
    }

	printf("x length by line counter: %d \n", line_counter);

    for(i=0; i < h_x_length; ++i){ 
        switch(test_case) {
            case 1:
                h_Y[i] = (0.2*(h_X[i*h_x_dim + 1] - 0.5)) +
                    (.6-h_X[i*h_x_dim + 2]) > 0 ? 1 : -1;
                break;
            case 2:
                h_Y[i] = (h_X[i*h_x_dim + 1]-.5)*(h_X[i*h_x_dim + 1]-.5) +
                    (h_X[i*h_x_dim + 2]-.5)*(h_X[i*h_x_dim + 2]-.5) > 0.09 ? 1 : -1;
                break;
            case 3:
                h_Y[i] = 4*(h_X[i*h_x_dim + 1]-.5)*(h_X[i*h_x_dim + 1]-.5) +
                    (.2-h_X[i*h_x_dim + 2]) > 0 ? 1 : -1;
                break;
            default:
                h_Y[i] = 0;
        }
    }
float exec_times[5][2];
int k, j;
float acc;;
int index = 0;
float current_eta = eta;
for(k = 0; k < 5; k++){
	for(i=0;i< h_x_length;i++){
		h_misclassified[i] = 1;
		h_not_classified[i] = 0;
		h_sum_missed[i] = 0;
	}
	for(i=0; i < h_x_dim; i++){
		h_W[i] = 0;
	}
	int h_W_size = NUM_BLOCKS* h_x_dim;
	for(i = 0; i < h_W_size; i++){
		h_W_matrix[i] = 0;
	}
	CUDA_SAFE_CALL(hipMemcpy(g_X, h_X, allocSize_X, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(g_Y, h_Y, allocSize_Y, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(g_W, h_W, allocSize_W, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(g_misclassified, h_misclassified, allocSize_Y, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(g_W_matrix, h_W_matrix, allocSize_W_mat, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(g_not_classified, h_not_classified,allocSize_sumMissed, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(g_sum_missed, h_sum_missed, allocSize_sumMissed, hipMemcpyHostToDevice)); 

	iters = 0;
	missed = 0;
	not_classified = 1;
	#ifdef PRINT_TIME
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	#endif

dim3 dimBlock(NUM_THREADS_PER_BLOCK, 1, 1);
dim3 dimGrid(NUM_BLOCKS, 1);
while(not_classified && iters <= MAX_ITERS){
		// Increment iters
		iters++;
		// Set condition to zero (to avoid infinite while loop) and set it to one if there's an element that is misclassified
		not_classified = 0;
		// One block with 500 threads (one thread working on each row of data in X)
		calculate_weights<<<dimGrid, dimBlock>>>(g_X, g_Y,g_W_matrix,g_misclassified,h_x_length, h_x_dim, current_eta);
		CUDA_SAFE_CALL(hipPeekAtLastError());
		hipDeviceSynchronize();
		// Copy weight vector to host
		CUDA_SAFE_CALL(hipMemcpy(h_W_matrix, g_W_matrix, allocSize_W_mat, hipMemcpyDeviceToHost));
		for(i=0;i<h_x_dim;i++){
			acc =0;
			for(j=0;j<NUM_BLOCKS;j++){
				acc += h_W_matrix[j*h_x_dim + i];
			}
			h_W[i] += acc;
		}
		hipDeviceSynchronize();
		CUDA_SAFE_CALL(hipMemcpy(g_W, h_W, allocSize_W, hipMemcpyHostToDevice));
		// Check classification success	
		hipDeviceSynchronize();	
		classify<<<dimGrid, dimBlock>>>(g_X, g_Y, g_W, g_misclassified, g_not_classified, g_sum_missed,h_x_dim);
		CUDA_SAFE_CALL(hipPeekAtLastError());
		hipDeviceSynchronize();
		// Copy arrays back to host
		CUDA_SAFE_CALL(hipMemcpy(h_not_classified, g_not_classified,allocSize_sumMissed, hipMemcpyDeviceToHost));
		CUDA_SAFE_CALL(hipMemcpy(h_sum_missed, g_sum_missed,allocSize_sumMissed, hipMemcpyDeviceToHost));
		for(i=0;i<h_x_length;i++){
			not_classified += h_not_classified[i];
		}
}
	

	#ifdef PRINT_TIME
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_gpu, start, stop);
	exec_times[k][0] = elapsed_gpu;
	printf("Exec time: %f \n", elapsed_gpu);
	hipEventDestroy(start);
	#endif
	printf("\n");
		for(i=0;i < h_x_length; ++i){
			missed += h_sum_missed[i];
		}
		if(missed == 0){
			printf("Perfectly separated data\n");
		}
		else{
			printf("Finished MAX_ITERS (%d iters) and still %d misclassified \n", iters, missed);
		}

	sum_missed_iters[k][0] = missed;
	sum_missed_iters[k][1] = iters;
	total_missed =0;
	for(i = 0; i < h_x_length; i++){
		verify = 0;
		for(j=0;j < h_x_dim; j++){
			verify += h_X[i*h_x_dim + j] * h_W[j];
		}
		verify = verify * h_Y[i];
		if(verify < 0){
			total_missed++;
		}
	}
	exec_times[k][1] = total_missed;
}
	printf("Verification Missed:	Exec time (ms)		Sum Missed: 		Iters: \n");
	int avgIters = 0;
	float avg_exec_time = 0;
	total_missed = 0;
	for(i=0;i<5; i++){
		avgIters += sum_missed_iters[i][1];
		avg_exec_time += exec_times[i][0];
		total_missed += sum_missed_iters[i][0];
		printf("\t%f\t\t\t%f\t\t%d\t\t\t%d\n", exec_times[i][1], exec_times[i][0], sum_missed_iters[i][0], sum_missed_iters[i][1]);
	}
	avgIters = avgIters/ 5;
	avg_exec_time = avg_exec_time/ 5;

	printf("Avg exec time: %f avg iters: %d sum missed (should be 0): %d \n", avg_exec_time, avgIters, total_missed);
		// Free-up device and host memory
	CUDA_SAFE_CALL(hipFree(g_X));
	CUDA_SAFE_CALL(hipFree(g_Y));
	CUDA_SAFE_CALL(hipFree(g_W));
	CUDA_SAFE_CALL(hipFree(g_sum_missed));
	CUDA_SAFE_CALL(hipFree(g_not_classified));
	CUDA_SAFE_CALL(hipFree(g_score));
	CUDA_SAFE_CALL(hipFree(g_misclassified));		   
	free(h_X);
	free(h_not_classified);
	free(h_sum_missed);
	free(h_Y);
	free(h_W);
	free(h_misclassified);
	free(h_score);
	return 0;
}

const char* getfield(char* line, int num) {
    const char* tok;
    for (tok = strtok(line, ",");
            tok && *tok;
            tok = strtok(NULL, ",\n"))
    {
        if (!--num)
            return tok;
    }
    return NULL;
}
