#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <math.h>
// Assertion to check for errors
#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"CUDA_SAFE_CALL: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

#define NUM_THREADS_PER_BLOCK 	500	
#define NUM_BLOCKS 		1	
#define PRINT_TIME		1
#define TEST_CASE		3
#define X_DIM                   6
#define X_LENGTH                500
#define START_ETA		0.1
#define ETA   			1.0
#define DELTA			.05
#define MAX_ITERS		10000
#define IMUL(a, b) __mul24(a, b)


const char* getfield(char* line, int num);

__global__ void calculate_weights(float* X, char* Y, float* W, char* misclassified,int x_length, int x_dim, double eta){
	__shared__ float block_weights[NUM_THREADS_PER_BLOCK][X_DIM]; // 500 x 6
	int tx = threadIdx.x;
	int i,j;
	
		if(misclassified[tx] == 1){
			for(j= 0; j < x_dim;j++){
				block_weights[tx][j] = eta*X[tx*x_dim+j]*Y[tx];
			}	
		}
		else{
			for(j=0; j < x_dim; j++){
				block_weights[tx][j] = 0;
			}
		}
	__syncthreads();

	float sum;
	if(tx==399){
		for(j=0;j<x_dim;j++){
			sum = 0;
			for(i=0; i < NUM_THREADS_PER_BLOCK;i++){
				sum = sum+  block_weights[i][j];
			}
			W[j]+= sum;
		}
	}
}

__global__ void classify(float* X, char* Y, float* W, char* misclassified, char* not_classified, int* sum_missed,  int x_dim){
	__shared__  float score_shared[NUM_THREADS_PER_BLOCK];
	int tx = threadIdx.x;
	int j;
	score_shared[tx] =0;
	sum_missed[tx] = 0;
	not_classified[tx] = 0;
	for(j=0;j < x_dim; ++j){
		score_shared[tx] += X[tx*x_dim + j]*W[j];
	}
	__syncthreads();
	misclassified[tx] = score_shared[tx]*Y[tx] <= 0.0 ? 1:0;
	__syncthreads();
	if(misclassified[tx] == 1){
		sum_missed[tx] = 1;	
		not_classified[tx] = 1;
	}
	__syncthreads();
}


int main(int argc, char **argv){
	// GPU Timing variables
	hipEvent_t start, stop;
	float elapsed_gpu;
	int test_case = TEST_CASE;
	int h_x_length = X_LENGTH;
	int h_x_dim = X_DIM;
	int line_counter = 0;
	int i;
	char not_classified = 1;
	int iters = 0;
	float eta = ETA;
	float start_eta = START_ETA;
	float delta = DELTA;
	int sum_missed_iters[19][2];
	// global variables on GPU
	float* g_W;			
	float* g_X;
	float* g_score;
	char* g_Y;
	char* g_not_classified;
	char* g_misclassified;
	int* g_sum_missed;


	//global arrays on host
	float* h_W;
	float* h_X;
	char* h_Y;
	float* h_score;
	char* h_misclassified;	
	char* h_not_classified;
	int* h_sum_missed;
	int missed = 0;

	 // Select GPU
	CUDA_SAFE_CALL(hipSetDevice(1));

	// Allocate GPU memory
	size_t allocSize_X =  h_x_dim * h_x_length * sizeof(float);
	size_t allocSize_Y = h_x_length * sizeof(char);
	size_t allocSize_W = h_x_dim * sizeof(float);
	size_t allocSize_Score = h_x_length * sizeof(float);
	size_t allocSize_sumMissed = sizeof(int)*h_x_length;

	CUDA_SAFE_CALL(hipMalloc((void **)&g_W, allocSize_W))
	CUDA_SAFE_CALL(hipMalloc((void **)&g_X, allocSize_X));
	CUDA_SAFE_CALL(hipMalloc((void **)&g_Y, allocSize_Y));
	CUDA_SAFE_CALL(hipMalloc((void **)&g_score, allocSize_Score));
	CUDA_SAFE_CALL(hipMalloc((void **)&g_misclassified, allocSize_Y));	
	CUDA_SAFE_CALL(hipMalloc((void **)&g_sum_missed, allocSize_sumMissed));
	CUDA_SAFE_CALL(hipMalloc((void **)&g_not_classified, allocSize_Y));

	// Allocate arrays on host memory
	h_X                     = (float *) malloc(allocSize_X);
	h_Y                   	= (char *) malloc(allocSize_Y);
	h_W              	= (float *) malloc(allocSize_W);
	h_misclassified 	= (char *) malloc(allocSize_Y);
	h_score			= (float *) malloc(allocSize_Score);
	h_sum_missed		= (int *) malloc(allocSize_sumMissed);
	h_not_classified 	= (char *) malloc(allocSize_Y);

	for(i=0;i< h_x_length;i++){
		h_misclassified[i] = 1;
	}
	for(i=0; i < h_x_dim; i++){
		h_W[i] = 0;
	}

    FILE* stream = fopen("data.csv", "r");

    char line[1024];
    while (fgets(line, 1024, stream)&&line_counter < 500)
    {
        char* tmp = strdup(line);
        int idx = line_counter*h_x_dim;
        h_X[idx] = 1.0;
        h_X[idx + 1] = strtod(getfield(tmp, 1), NULL);
        tmp = strdup(line);
        h_X[idx + 2] = strtod(getfield(tmp, 2), NULL);
        h_X[idx + 3] = h_X[idx + 1]*h_X[idx + 2]; // xy
        h_X[idx + 4] = h_X[idx + 1]*h_X[idx + 1]; // x^2 
        h_X[idx + 5] = h_X[idx + 2]*h_X[idx + 2]; // y^2
        // NOTE strtok clobbers tmp
        free(tmp);
        line_counter++;
    }

//    assign_labels(h_X, h_x_length, h_x_dim, test_case, h_Y);

    for(i=0; i < h_x_length; ++i){ 
        switch(test_case) {
            case 1:
                h_Y[i] = (0.2*(h_X[i*h_x_dim + 1] - 0.5)) +
                    (.6-h_X[i*h_x_dim + 2]) > 0 ? 1 : -1;
                break;
            case 2:
                h_Y[i] = (h_X[i*h_x_dim + 1]-.5)*(h_X[i*h_x_dim + 1]-.5) +
                    (h_X[i*h_x_dim + 2]-.5)*(h_X[i*h_x_dim + 2]-.5) > 0.09 ? 1 : -1;
                break;
            case 3:
                h_Y[i] = 4*(h_X[i*h_x_dim + 1]-.5)*(h_X[i*h_x_dim + 1]-.5) +
                    (.2-h_X[i*h_x_dim + 2]) > 0 ? 1 : -1;
                break;
            default:
                h_Y[i] = 0;
        }
    }
int j;
 printf("X & Y : \n");
for(i = 0; i < h_x_length; i++){
	for(j= 0; j < h_x_dim; j++){
		printf("%f ", h_X[i*h_x_dim + j]);
	}
	printf("%f\n ", h_Y[i]);
}


    // Transfer the arrays to the GPU memory
	/*CUDA_SAFE_CALL(hipMemcpy(g_X, h_X, allocSize_X, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(g_Y, h_Y, allocSize_Y, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(g_W, h_W, allocSize_W, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(g_misclassified, h_misclassified, allocSize_Y, hipMemcpyHostToDevice));
*/
/*#ifdef PRINT_TIME
hipEventCreate(&start);
hipEventCreate(&stop);
hipEventRecord(start,0);
#endif*/
float exec_times[19][2];
//int num_blocks = NUM_BLOCKS;
//int num_threads = NUM_THREADS_PER_BLOCK;
int k;
int index = 0;
float current_eta = start_eta;
for(k = 0; k < 19; k++){
	for(i=0;i< h_x_length;i++){
		h_misclassified[i] = 1;
	}
	for(i=0; i < h_x_dim; i++){
		h_W[i] = 0;
	}
	CUDA_SAFE_CALL(hipMemcpy(g_X, h_X, allocSize_X, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(g_Y, h_Y, allocSize_Y, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(g_W, h_W, allocSize_W, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(g_misclassified, h_misclassified, allocSize_Y, hipMemcpyHostToDevice));
	iters = 0;
	missed = 0;
	not_classified = 1;
	#ifdef PRINT_TIME
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	#endif
	int max_iters = MAX_ITERS;


while(not_classified && iters <= MAX_ITERS){
		// Increment iters
		iters++;
		// Set condition to zero (to avoid infinite while loop) and set it to one if there's an element that is misclassified
		not_classified = 0;
		// One block with 500 threads (one thread working on each row of data in X)
		calculate_weights<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK>>>(g_X, g_Y,g_W,g_misclassified,h_x_length, h_x_dim, current_eta);
		CUDA_SAFE_CALL(hipPeekAtLastError());
		hipDeviceSynchronize();
		// Copy weight vector to host
		CUDA_SAFE_CALL(hipMemcpy(h_W, g_W, allocSize_W, hipMemcpyDeviceToHost));
		// Check classification success		
		classify<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK>>>(g_X, g_Y, g_W, g_misclassified, g_not_classified, g_sum_missed,h_x_dim);
		CUDA_SAFE_CALL(hipPeekAtLastError());
		hipDeviceSynchronize();
		// Copy arrays back to host
		CUDA_SAFE_CALL(hipMemcpy(h_not_classified, g_not_classified,allocSize_Y, hipMemcpyDeviceToHost));
		CUDA_SAFE_CALL(hipMemcpy(h_sum_missed, g_sum_missed,allocSize_sumMissed, hipMemcpyDeviceToHost));
		for(i=0;i<h_x_length;i++){
			not_classified += h_not_classified[i];		
		}
}
	

	#ifdef PRINT_TIME
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_gpu, start, stop);
	exec_times[index][1] = elapsed_gpu;
	exec_times[index][0] = current_eta;
	hipEventDestroy(start);
	#endif
	printf("\n");
		for(i=0;i < h_x_length; ++i){
			missed += h_sum_missed[i];
		}
		printf("current_eta: %f, eta: %f, start_eta: %f, index: %d, delta: %f \n", current_eta, eta, start_eta, index, delta);
		if(missed == 0){
			printf("Perfectly separated data\n");
		}
		else{
			printf("Finished MAX_ITERS (%d iters) and still %d misclassified \n", iters, missed);
		}
	sum_missed_iters[index][0] = missed;
	sum_missed_iters[index][1] = iters;
	printf("iters out of loop: %d", iters);
	current_eta += delta;
	index++;
}
	printf("Iters		Exec time (ms)		Sum Missed: 		Iters: \n");
	for(i=0;i<19; i++){
		printf("%f\t\t%f\t\t%d\t\t%d\n", exec_times[i][0], exec_times[i][1], sum_missed_iters[i][0], sum_missed_iters[i][1]);
	}
		// Free-up device and host memory
	CUDA_SAFE_CALL(hipFree(g_X));
	CUDA_SAFE_CALL(hipFree(g_Y));
	CUDA_SAFE_CALL(hipFree(g_W));
	CUDA_SAFE_CALL(hipFree(g_sum_missed));
	CUDA_SAFE_CALL(hipFree(g_not_classified));
	CUDA_SAFE_CALL(hipFree(g_score));
	CUDA_SAFE_CALL(hipFree(g_misclassified));		   
	free(h_X);
	free(h_not_classified);
	free(h_sum_missed);
	free(h_Y);
	free(h_W);
	free(h_misclassified);
	free(h_score);
	return 0;
}

const char* getfield(char* line, int num) {
    const char* tok;
    for (tok = strtok(line, ",");
            tok && *tok;
            tok = strtok(NULL, ",\n"))
    {
        if (!--num)
            return tok;
    }
    return NULL;
}
